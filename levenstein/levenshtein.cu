#include "hip/hip_runtime.h"
#include "levenshtein.h"
#include <hip/hip_runtime.h>
#include <math.h>

__device__ int __min(int a, int b) {
    return ((a)-(((a)-(b))&((b)-(a))>>31));
}

__device__ int __index(int i , int j, int n)
{
	int rval;
	if(!(i >= 0 && i <=n && j >= 0 && j <=n) ) {
		rval = 0;
	} else if((i+j) <= n) {
                rval = (((i+j)*(i+j+1))/2) + j;
	} else {
		rval = (n*n) - (((2*n - (i+j))*(2*n - (i+j+1)))/2) +
			(j - ((j+i) - n)) - 1;
	}
	return rval;
}

__global__ void levenshteinKernel(char* Md, char* Nd, int* Rd, int size) {
    __shared__ char Nds[ARRSIZE];   //Shared Nd character memory
    __shared__ int  Rs[ARRSIZE];    //Shared current min value memory
    int col = threadIdx.x + 1;      //column
    int row;                        //row
    char Mdt = Md[threadIdx.x];     //Character for this column

    Rd[0] = 0;
    Rd[__index(0, col,ARRSIZE)] = col;
    Rd[__index(col, 0,ARRSIZE)] = col;
    Nds[threadIdx.x]   = Nd[threadIdx.x];
    Rs[threadIdx.x]    = Rd[__index(0,col,ARRSIZE)];
    __syncthreads();

    for(int k = 2; k < (2 * size) + 1; ++k) { 
        row = k - threadIdx.x;
        if( row > 0 && row <= size)
        {
            Rs[threadIdx.x]       = __min( (Rd[__index(row-1,col,ARRSIZE)] + 1),
                                           (Rd[__index(row,col-1,ARRSIZE)] + 1 ) );
            Rd[__index(row,col,ARRSIZE)]  = __min( (Rs[threadIdx.x]),
                                           (Rd[__index(row-1,col-1,ARRSIZE)] + ((Mdt!=Nds[row-1])&1)) );
        }
        __syncthreads();
    }    
}

__host__ void levenshteinCuda(char* s1, char* s2, int* &result, size_t size) {
    //Assumption is made that the size is a multiple of tile size
    dim3 dimGrid(1, 1);
    dim3 dimBlock(ARRSIZE, 1);
    
    char* Sd;
    char* Td;
    int*  Rd;
    size_t arrSize = (ARRSIZE+1) * (ARRSIZE+1);
    Sd = Td = NULL;
    Rd = NULL;

    hipMalloc((void**) &Sd, (size *   sizeof(char)));
    hipMalloc((void**) &Td, (size *   sizeof(char)));
    hipMalloc((void**) &Rd, (arrSize *    sizeof(int)));

    hipMemcpy(Sd, s1,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Td, s2,     (size * sizeof(char)), hipMemcpyHostToDevice);

    levenshteinKernel<<<dimGrid, dimBlock>>>(Sd,Td,Rd,size);

    hipMemcpy(result, Rd, (arrSize * sizeof(int)), hipMemcpyDeviceToHost);

    hipFree(Sd);
    hipFree(Td);
    hipFree(Rd);
    return;
}

__host__ int getIndex(int row , int col)
{
    return ((row * (ARRSIZE + 1)) + col);
}

__host__ int getMin(int a, int b)
{
    return (a-((a-b)&(b-a)>>31));
}

