#include "hip/hip_runtime.h"
#include "levenshtein.h"
#include <hip/hip_runtime.h>
#include <math.h>


__device__ int __index(int i,int j) {
    return ((i * (ARRSIZE + 1)) + j);
}

__device__ int __min(int a, int b) {
    return (a-((a-b)&(b-a)>>31));
}


__global__ void levenshteinKernel(char* Md, char* Nd, int* Rd, int size) {
    __shared__ char Mds[ARRSIZE];   //Shared Md character memory
    __shared__ char Nds[ARRSIZE];   //Shared Nd character memory
    __shared__ int  Rs[ARRSIZE];    //Shared current min value memory
    //__shared__ int  Rprev[ARRSIZE]; //Shared top value memory

    Mds[threadIdx.x]   = Md[threadIdx.x];
    Nds[threadIdx.x]   = Nd[threadIdx.x];
    Rs[threadIdx.x]    = Rd[threadIdx.x];
    //Rprev[threadIdx.x] = Rs[threadIdx.x];

    __syncthreads();

    int i = threadIdx.x + 1;  //column
    int j;                    //row

    for(int k = 2; k < (2 * size) - 1; ++k) {
        j = k - threadIdx.x - 1;
        if( j > 0 && j < size)
        {
            Rs[threadIdx.x]   = __min( (Rd[__index(i,j-1)] + 1),
                                       (Rd[__index(i-1,j)] + 1 )   );
            Rd[__index(i,j)]  = __min( (Rs[threadIdx.x]),
                                       (Rd[__index(i-1,j-1)] + ((Mds[i-1]!=Nds[j-1])&1)) );
            Rs[threadIdx.x] = Rd[__index(i,j)];
        }

        __syncthreads();
        //Rprev[threadIdx.x] = Rs[threadIdx.x];
        //__syncthreads();
    }    
}

__host__ void levenshteinCuda(char* s1, char* s2, int* &result, size_t size) {
    //Assumption is made that the size is a multiple of tile size
    dim3 dimGrid(1, 1);
    dim3 dimBlock(ARRSIZE, 1);
    
    char* Sd;
    char* Td;
    int*  Rd;
    size_t arrSize = (ARRSIZE+1) * (ARRSIZE+1);
    Sd = Td = NULL;
    Rd = NULL;

    for(int i = 0; i <= ARRSIZE; ++i) //for each element in the first column
        result[getIndex(i,0)] = i;

    for (int i = 0; i <= ARRSIZE; i++)
        result[getIndex(0,i)] = i;

    hipMalloc((void**) &Sd, (size *   sizeof(char)));
    hipMalloc((void**) &Td, (size *   sizeof(char)));
    hipMalloc((void**) &Rd, (arrSize *    sizeof(int)));

    hipMemcpy(Sd, s1,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Td, s2,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Rd, result, (arrSize * sizeof(int)),  hipMemcpyHostToDevice);
   
    levenshteinKernel<<<dimGrid, dimBlock>>>(Sd,Td,Rd,size);

    hipMemcpy(result, Rd, (arrSize * sizeof(int)), hipMemcpyDeviceToHost);

    hipFree(Sd);
    hipFree(Td);
    hipFree(Rd);



    return;
}

__host__ int getIndex(int i , int j)
{
    return ((i * (ARRSIZE + 1)) + j);
}

__host__ int getMin(int a, int b)
{
    return (a-((a-b)&(b-a)>>31));
}