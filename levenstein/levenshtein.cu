#include "hip/hip_runtime.h"
#include "levenshtein.h"
#include <hip/hip_runtime.h>
#include <math.h>

#define index(i,j) ((i * ARRSIZE) + j)


__global__ void levenshteinKernel(char* Md, char* Nd, int* Rd, int size) {
    __shared__ char Mds[ARRSIZE];   //Shared Md character memory
    __shared__ char Nds[ARRSIZE];   //Shared Nd character memory
    __shared__ int  Rs[ARRSIZE];    //Shared current min value memory
    __shared__ int  Rprev[ARRSIZE]; //Shared top value memory

    Mds[threadIdx.x]   = Md[threadIdx.x];
    Nds[threadIdx.x]   = Nd[threadIdx.x];
    Rs[threadIdx.x]    = Rd[threadIdx.x];
    Rprev[threadIdx.x] = Rs[threadIdx.x];

    __syncthreads();

    int i = threadIdx.x + 1;  //column
    int j;                    //row

    for(int k = 0; k < (2 * size) - 1; ++k) {
        j = k - threadIdx.x;
        if( j >=0 && j < size)
        {
            Rs[threadIdx.x] = MIN( (Rd[index(j, i - 1)] + 1),
                                   (Rprev[threadIdx.x] + 1) );
            Rd[index(j,i)]  = MIN( (Rs[threadIdx.x]),
                                   (Rd[index(j-1,i-1)] + ((Mds[i-1]==Nds[j-1])&1)) );
        }

        __syncthreads();
        Rprev[threadIdx.x] = Rs[threadIdx.x];
        __syncthreads();
    }
    
    
}

__host__ void levenshteinCuda(char* s1, char* s2, int** &result,
        size_t size) {
    //Assumption is made that the size is a multiple of tile size
    dim3 dimGrid(1, 1);
    dim3 dimBlock(ARRSIZE, 1);
    
    char* Sd;
    char* Td;
    int*  Rd;
    size_t arrSize = ARRSIZE * ARRSIZE;
    Sd = Td = NULL;
    Rd = NULL;

    for (int i = 0; i < ARRSIZE + 1; i++)
        result[i][0] = i;

    for (int i = 0; i < ARRSIZE + 1; i++)
        result[0][i] = i;

    hipMalloc((void**) &Sd, (size *   sizeof(char)));
    hipMalloc((void**) &Td, (size *   sizeof(char)));
    hipMalloc((void**) &Rd, (arrSize *    sizeof(int)));

    hipMemcpy(Sd, s1,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Td, s2,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Rd, result, (arrSize * sizeof(int)),  hipMemcpyHostToDevice);
   
    levenshteinKernel<<<dimGrid, dimBlock>>>(Sd,Td,Rd,size);

    hipMemcpy(result, Rd, (arrSize * sizeof(size_t)), hipMemcpyDeviceToHost);

    hipFree(Sd);
    hipFree(Td);
    hipFree(Rd);

    return;
}