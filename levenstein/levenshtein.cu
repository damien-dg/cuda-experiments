#include "hip/hip_runtime.h"
#include "levenshtein.h"
#include <hip/hip_runtime.h>
#include <math.h>


__device__ int __index(int row,int col) {
    return (((row) * (ARRSIZE + 1)) + (col));
}

__device__ int __min(int a, int b) {
    return ((a)-(((a)-(b))&((b)-(a))>>31));
}

__device__ int tiledIndex(int row, int column, int width) {
    return (((((row<=width)&&(row>0))&&((col<=width)&&(col>0)))*0xffffffff)& \
            (((((row+col)*(row+col+1))/2)+col+1)));
}


__global__ void levenshteinKernel(char* Md, char* Nd, int* Rd, int size) {
    __shared__ char Nds[ARRSIZE];   //Shared Nd character memory
    __shared__ int  Rs[ARRSIZE];    //Shared current min value memory
    int col = threadIdx.x + 1;      //column
    int row;                        //row
    char Mdt = Md[threadIdx.x];     //Character for this column
    
    Nds[threadIdx.x]   = Nd[threadIdx.x];
    Rs[threadIdx.x]    = Rd[__index(0,col)];
    __syncthreads();

    for(int k = 2; k < (2 * size) + 1; ++k) { 
        row = k - threadIdx.x;
        if( row > 0 && row <= size)
        {
            Rs[threadIdx.x]       = __min( (Rd[__index(row-1,col)] + 1),
                                           (Rd[__index(row,col-1)] + 1 ) );
            Rd[__index(row,col)]  = __min( (Rs[threadIdx.x]),
                                           (Rd[__index(row-1,col-1)] + ((Mdt!=Nds[row-1])&1)) );
        }
        __syncthreads();
    }    
}

__host__ void levenshteinCuda(char* s1, char* s2, int* &result, size_t size) {
    //Assumption is made that the size is a multiple of tile size
    dim3 dimGrid(1, 1);
    dim3 dimBlock(ARRSIZE, 1);
    
    char* Sd;
    char* Td;
    int*  Rd;
    size_t arrSize = (ARRSIZE+1) * (ARRSIZE+1);
    Sd = Td = NULL;
    Rd = NULL;

    for(int i = 0; i <= ARRSIZE; ++i) //for each element in the first column
        result[getIndex(i,0)] = i;

    for (int i = 0; i <= ARRSIZE; i++)
        result[getIndex(0,i)] = i;

    hipMalloc((void**) &Sd, (size *   sizeof(char)));
    hipMalloc((void**) &Td, (size *   sizeof(char)));
    hipMalloc((void**) &Rd, (arrSize *    sizeof(int)));

    hipMemcpy(Sd, s1,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Td, s2,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Rd, result, (arrSize * sizeof(int)),  hipMemcpyHostToDevice);
#ifdef TESTING
    for( int z = 0; z < TESTLENGTH; ++z) {
#endif
        levenshteinKernel<<<dimGrid, dimBlock>>>(Sd,Td,Rd,size);
#ifdef TESTING
    }
#endif

    hipMemcpy(result, Rd, (arrSize * sizeof(int)), hipMemcpyDeviceToHost);

    hipFree(Sd);
    hipFree(Td);
    hipFree(Rd);
    return;
}

__host__ int getIndex(int row , int col)
{
    return ((row * (ARRSIZE + 1)) + col);
}

__host__ int getMin(int a, int b)
{
    return (a-((a-b)&(b-a)>>31));
}

__host__ int getTiledIndex(int row, int col, int width)
{
    return (((((row<=width)&&(row>0))&&((col<=width)&&(col>0)))*0xffffffff)& \
            (((((row+col)*(row+col+1))/2)+col+1)));
}