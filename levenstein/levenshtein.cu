#include "hip/hip_runtime.h"
#include "levenshtein.h"
#include <hip/hip_runtime.h>
#include <math.h>

__device__ int __min(int a, int b) {
    return ((a)-(((a)-(b))&((b)-(a))>>31));
}

//N must be the size of the array
__device__ int __index(int i , int j, int n)
{
	int rval;
	if(!(i >= 0 && i < (n) && j >= 0 && j < (n) ) ) {
		rval = 0;
	} else if((i+j) < (n)) {
                rval = (((i+j)*(i+j+1))/2) + j;
	} else {
		rval = ((n)*(n)) -
                        (((2*(n) - (i+j))*(2*(n) - (i+j+1)))/2) +
			(j - ((j+i) - (n))) - 1;
	}
	return rval;
}

__global__ void levenshteinKernel(char* Md, char* Nd, int* Rd, int size) {
    __shared__ char Nds[ARRSIZE];   //Shared Nd character memory
    __shared__ int  Rs[ARRSIZE];    //Shared current min value memory
    int col = threadIdx.x + 1;      //column
    int row;                        //row
    char Mdt = Md[threadIdx.x];     //Character for this column

    Rd[0] = 0;

    Rd[__index(0, col,size)] = col;
    Rd[__index(col, 0,size)] = col;
    
    Nds[threadIdx.x]   = Nd[threadIdx.x];
    Rs[threadIdx.x]    = Rd[__index(0,col,size)];
    __syncthreads();

    
    for(int k = 2; k < (2 * size) + 1; ++k) {
        row = k - threadIdx.x;
        if( row > 0 && row <= size && col > 0 && col <= size )
        {
            Rs[threadIdx.x]       = __min(
                    (Rd[__index(row-1,col,size)] + 1),
                    (Rd[__index(row,col-1,size)] + 1 ) );
            Rd[__index(row,col,size)]  = __min(
                    (Rs[threadIdx.x]),
                    (Rd[__index(row-1,col-1,size)] + ((Mdt!=Nds[row-1])&1)) );
        }
        __syncthreads();
    }
}

__host__ void levenshteinCuda(char* s1, char* s2, int* &result, size_t size) {
    //Assumption is made that the size is a multiple of tile size
    dim3 dimGrid(1, 1);
    dim3 dimBlock(size, 1);
    
    char* Sd;
    char* Td;
    int*  Rd;
    size_t arrSize = (size+1) * (size+1);
    Sd = Td = NULL;
    Rd = NULL;

    hipMalloc((void**) &Sd, (size *   sizeof(char)));
    hipMalloc((void**) &Td, (size *   sizeof(char)));
    hipMalloc((void**) &Rd, (arrSize *    sizeof(int)));

    hipMemcpy(Sd, s1,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemcpy(Td, s2,     (size * sizeof(char)), hipMemcpyHostToDevice);
    hipMemset(Rd, 0, arrSize * sizeof(int));

    levenshteinKernel<<<dimGrid, dimBlock>>>(Sd,Td,Rd,size+1);

    hipMemcpy(result, Rd, (arrSize * sizeof(int)), hipMemcpyDeviceToHost);

    hipFree(Sd);
    hipFree(Td);
    hipFree(Rd);
    return;
}

__host__ int getIndex(int row , int col)
{
    return ((row * (ARRSIZE + 1)) + col);
}

__host__ int getMin(int a, int b)
{
    return (a-((a-b)&(b-a)>>31));
}

